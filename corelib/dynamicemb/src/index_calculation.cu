#include "hip/hip_runtime.h"
/******************************************************************************
# SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
# SPDX-License-Identifier: Apache-2.0
#
# Licensed under the Apache License, Version 2.0 (the "License");
# you may not use this file except in compliance with the License.
# You may obtain a copy of the License at
#
# http://www.apache.org/licenses/LICENSE-2.0
#
# Unless required by applicable law or agreed to in writing, software
# distributed under the License is distributed on an "AS IS" BASIS,
# WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
# See the License for the specific language governing permissions and
# limitations under the License.
******************************************************************************/

#include "check.h"
#include "index_calculation.h"
#include <torch/extension.h>
#include <cuda/std/tuple>
#include <iostream>
#include <type_traits>
namespace { // anonymous namespace

template <typename T>
HOST_DEVICE_INLINE int64_t bs_upper_bound_sub_one(const T *const arr,
                                                  int64_t num, T target) {
  int64_t start = 0;
  int64_t end = num;
  while (start < end) {
    int64_t middle = start + (end - start) / 2;
    T value = arr[middle];
    if (value <= target) {
      start = middle + 1;
    } else {
      end = middle;
    }
  }
  return (start == num && arr[start - 1] != target) ? num : start - 1;
}

template <typename key_t, typename seg_id_t = int32_t> struct SegmentedKey {
  struct Decomposer {
    __host__ __device__ cuda::std::tuple<seg_id_t &, key_t &>
    operator()(SegmentedKey<key_t, seg_id_t> &segmented_key) const {
      return {segmented_key.segment_id, segmented_key.key};
    }
  };
  seg_id_t segment_id;
  key_t key;
} __attribute__((packed));

template <typename key_t, typename seg_id_t,
          typename ComposeKey = SegmentedKey<key_t, seg_id_t>>
__global__ void decompose_segmented_key_kernel(
    const ComposeKey *__restrict__ compose_arr, key_t *__restrict__ keys_out,
    seg_id_t *__restrict__ segment_ids_out, const int64_t num_keys) {
  CUDA_1D_KERNEL_LOOP(tid, num_keys) {
    ComposeKey compose = compose_arr[tid];
    keys_out[tid] = compose.key;
    if (segment_ids_out != nullptr) {
      segment_ids_out[tid] = compose.segment_id;
    }
  }
}

template <typename key_t, typename value_t, typename seg_id_t = int32_t,
          typename id_t = value_t,
          typename ComposeKey = SegmentedKey<key_t, seg_id_t>>
__global__ void segmented_sort_input_init_kernel(
    const key_t *__restrict__ keys, value_t *__restrict__ values,
    const id_t *__restrict__ segment_offsets, const int64_t num_keys,
    const int32_t num_segment_offsets, ComposeKey *__restrict__ compose_arr,
    seg_id_t *__restrict__ segment_ids_out) {
  CUDA_1D_KERNEL_LOOP(tid, num_keys) {
    key_t tmp_key = keys[tid];
    if (values != nullptr) {
      values[tid] = static_cast<value_t>(tid);
    }
    seg_id_t segment_id = bs_upper_bound_sub_one(
        segment_offsets, num_segment_offsets, static_cast<id_t>(tid));
    if (compose_arr != nullptr) {
      ComposeKey compose;
      compose.segment_id = segment_id;
      compose.key = tmp_key;
      compose_arr[tid] = compose;
    }
    if (segment_ids_out != nullptr) {
      segment_ids_out[tid] = segment_id;
    }
  }
}

template <typename key_t, typename seg_id_t>
__global__ void set_keys_flag(const key_t *__restrict__ sorted_keys,
                              const seg_id_t *__restrict__ sorted_segment_ids,
                              uint32_t *__restrict__ key_flag_buffer,
                              const int64_t num_key) {
  CUDA_1D_KERNEL_LOOP(tid, num_key) {
    key_t local_key = sorted_keys[tid];
    seg_id_t segment_id = sorted_segment_ids[tid];
    uint32_t is_first = 0;
    if ((tid == 0) ||
        ((tid > 0) && ((sorted_keys[tid - 1] != local_key) ||
                       (sorted_segment_ids[tid - 1] != segment_id)))) {
      is_first = 1;
    }
    key_flag_buffer[tid] = is_first;
  }
}

/// TODO:optimize with reschedule
template <typename key_t, typename id_t>
__global__ void set_unique_keys_and_unique_ids(
    const key_t *__restrict__ sorted_keys,
    const uint32_t
        *__restrict__ key_flag_buffer, /// TODO:upgrade to uint64_t/int64_t
    key_t *__restrict__ unique_keys, id_t *__restrict__ unique_key_ids,
    const int64_t key_num) {
  CUDA_1D_KERNEL_LOOP(tid, key_num) {
    uint32_t key_buffer = key_flag_buffer[tid];
    unique_key_ids[tid] = key_buffer - 1;
    if ((tid > 0 && key_flag_buffer[tid - 1] != key_buffer) || tid == 0) {
      unique_keys[key_buffer - 1] = sorted_keys[tid];
    }
  }
}

} // anonymous namespace

namespace dyn_emb {

SegmentedSortDevice::SegmentedSortDevice(c10::Device &device, int64_t key_num,
                                         int32_t segment_num, DataType key_type,
                                         DataType value_type)
    : device_(device), key_num_(key_num), segment_num_(segment_num),
      key_type_(key_type), value_type_(value_type) {

#if CUB_VERSION >= 200200
  need_compose_flag_ = true;
  set_out_segment_ids_ = false;
  DISPATCH_INTEGER_DATATYPE_FUNCTION(key_type_, key_t, [&] {
    DISPATCH_INTEGER_DATATYPE_FUNCTION(value_type_, value_t, [&] {
      using ComposeKey = SegmentedKey<key_t, seg_id_t>;
      this->segmented_keys_in_ =
          at::empty({static_cast<int64_t>(key_num_ * (sizeof(ComposeKey)))},
                    at::TensorOptions().dtype(torch::kChar).device(device_));
      this->segmented_keys_out_ =
          at::empty({static_cast<int64_t>(key_num_ * (sizeof(ComposeKey)))},
                    at::TensorOptions().dtype(torch::kChar).device(device_));
      hipcub::DeviceRadixSort::SortPairs<ComposeKey, value_t>(
          nullptr, cub_sort_temp_bytes_, nullptr, nullptr, nullptr, nullptr,
          key_num_, ComposeKey::Decomposer{}, 0, sizeof(ComposeKey) * 8);
      cub_sort_temp_buffer_ =
          at::empty({static_cast<int64_t>(cub_sort_temp_bytes_)},
                    at::TensorOptions().dtype(torch::kChar).device(device_));
    });
  });
#else
  need_compose_flag_ = false;
  set_out_segment_ids_ = true;
#endif
}

void SegmentedSortDevice::operator()(
    const at::Tensor &keys_in, at::Tensor &values_in,
    const at::Tensor &segment_offsets, at::Tensor &keys_out,
    at::Tensor &values_out, at::Tensor &segment_ids_out, hipStream_t &stream,
    bool set_input_value_to_idx, bool set_output_segment_ids) {
  if (key_num_ == 0)
    return;
  /// TODO: to optimize the grid_size
  auto &device_prop = DeviceProp::getDeviceProp(keys_in.device().index());
  constexpr int block_size = 256;
  const int grid_size = device_prop.total_threads / block_size;

  // initialize input.
  DISPATCH_INTEGER_DATATYPE_FUNCTION(key_type_, key_t, [&] {
    DISPATCH_INTEGER_DATATYPE_FUNCTION(value_type_, value_t, [&] {
      using ComposeKey = SegmentedKey<key_t, seg_id_t>;
      segmented_sort_input_init_kernel<key_t, value_t, seg_id_t, value_t>
          <<<grid_size, block_size, 0, stream>>>(
              reinterpret_cast<key_t *>(keys_in.data_ptr()),
              set_input_value_to_idx
                  ? reinterpret_cast<value_t *>(values_in.data_ptr())
                  : nullptr,
              reinterpret_cast<value_t *>(segment_offsets.data_ptr()), key_num_,
              segment_num_ + 1,
              need_compose_flag_ ? reinterpret_cast<ComposeKey *>(
                                       segmented_keys_in_.data_ptr())
                                 : nullptr,
              set_out_segment_ids_
                  ? reinterpret_cast<seg_id_t *>(segment_ids_out.data_ptr())
                  : nullptr);
    });
  });
  DEMB_CUDA_KERNEL_LAUNCH_CHECK();
  // sort
#if CUB_VERSION >= 200200
  DISPATCH_INTEGER_DATATYPE_FUNCTION(key_type_, key_t, [&] {
    DISPATCH_INTEGER_DATATYPE_FUNCTION(value_type_, value_t, [&] {
      using ComposeKey = SegmentedKey<key_t, seg_id_t>;
      hipcub::DeviceRadixSort::SortPairs(
          cub_sort_temp_buffer_.data_ptr(), cub_sort_temp_bytes_,
          reinterpret_cast<ComposeKey *>(segmented_keys_in_.data_ptr()),
          reinterpret_cast<ComposeKey *>(segmented_keys_out_.data_ptr()),
          reinterpret_cast<value_t *>(values_in.data_ptr()),
          reinterpret_cast<value_t *>(values_out.data_ptr()), key_num_,
          ComposeKey::Decomposer{}, 0, sizeof(ComposeKey) * 8, stream);
      decompose_segmented_key_kernel<<<grid_size, block_size, 0, stream>>>(
          reinterpret_cast<ComposeKey *>(segmented_keys_out_.data_ptr()),
          reinterpret_cast<key_t *>(keys_out.data_ptr()),
          set_output_segment_ids
              ? reinterpret_cast<seg_id_t *>(segment_ids_out.data_ptr())
              : nullptr,
          key_num_);
    });
  });
  DEMB_CUDA_KERNEL_LAUNCH_CHECK();
#else
  DISPATCH_INTEGER_DATATYPE_FUNCTION(key_type_, key_t, [&] {
    DISPATCH_INTEGER_DATATYPE_FUNCTION(value_type_, value_t, [&] {
      void *dummy_ptr = nullptr;
      hipcub::DeviceSegmentedRadixSort::SortPairs<key_t, value_t>(
          dummy_ptr, cub_sort_temp_bytes_,
          reinterpret_cast<key_t *>(keys_in.data_ptr()),
          reinterpret_cast<key_t *>(keys_out.data_ptr()),
          reinterpret_cast<value_t *>(values_in.data_ptr()),
          reinterpret_cast<value_t *>(values_out.data_ptr()), key_num_,
          segment_num_, reinterpret_cast<value_t *>(segment_offsets.data_ptr()),
          reinterpret_cast<value_t *>(segment_offsets.data_ptr()) + 1, 0,
          sizeof(key_t) * 8, stream);
      cub_sort_temp_buffer_ =
          at::empty({static_cast<int64_t>(cub_sort_temp_bytes_)},
                    at::TensorOptions().dtype(torch::kChar).device(device_));
      hipcub::DeviceSegmentedRadixSort::SortPairs(
          cub_sort_temp_buffer_.data_ptr(), cub_sort_temp_bytes_,
          reinterpret_cast<key_t *>(keys_in.data_ptr()),
          reinterpret_cast<key_t *>(keys_out.data_ptr()),
          reinterpret_cast<value_t *>(values_in.data_ptr()),
          reinterpret_cast<value_t *>(values_out.data_ptr()), key_num_,
          segment_num_, reinterpret_cast<value_t *>(segment_offsets.data_ptr()),
          reinterpret_cast<value_t *>(segment_offsets.data_ptr()) + 1, 0,
          sizeof(key_t) * 8, stream);
    });
  });
#endif
}

SegmentedUniqueDevice::SegmentedUniqueDevice(c10::Device &device,
                                             int64_t num_key, DataType key_type,
                                             DataType id_type)
    : device_(device), num_key_(num_key), key_type_(key_type),
      id_type_(id_type) {
  key_flag_buffer_ =
      at::empty({static_cast<int64_t>(num_key_)},
                at::TensorOptions().dtype(torch::kUInt32).device(device_));
  hipcub::DeviceScan::InclusiveSum<uint32_t *, uint32_t *>(
      nullptr, cub_scan_temp_bytes_, nullptr, nullptr, num_key_);
  cub_scan_temp_buffer_ =
      at::empty({static_cast<int64_t>(cub_scan_temp_bytes_)},
                at::TensorOptions().dtype(torch::kChar).device(device_));
}

void SegmentedUniqueDevice::operator()(
    const at::Tensor &sorted_keys, const at::Tensor &sorted_segment_ids,
    at::Tensor &unique_keys,
    at::Tensor &unique_key_ids, // mapping from sorted keys to unique keys
    hipStream_t &stream) {

  if (num_key_ == 0)
    return;
  auto &device_prop = DeviceProp::getDeviceProp(sorted_keys.device().index());
  const int block_size = 256;
  const int grid_size = device_prop.total_threads / block_size;

  DISPATCH_INTEGER_DATATYPE_FUNCTION(key_type_, key_t, [&] {
    set_keys_flag<key_t, seg_id_t><<<grid_size, block_size, 0, stream>>>(
        reinterpret_cast<key_t *>(sorted_keys.data_ptr()),
        reinterpret_cast<seg_id_t *>(sorted_segment_ids.data_ptr()),
        reinterpret_cast<uint32_t *>(key_flag_buffer_.data_ptr()), num_key_);
  });
  DEMB_CUDA_KERNEL_LAUNCH_CHECK();
  hipcub::DeviceScan::InclusiveSum(
      cub_scan_temp_buffer_.data_ptr(), cub_scan_temp_bytes_,
      reinterpret_cast<uint32_t *>(key_flag_buffer_.data_ptr()),
      reinterpret_cast<uint32_t *>(key_flag_buffer_.data_ptr()), num_key_,
      stream);
  DEMB_CUDA_KERNEL_LAUNCH_CHECK();
  DISPATCH_INTEGER_DATATYPE_FUNCTION(key_type_, key_t, [&] {
    DISPATCH_INTEGER_DATATYPE_FUNCTION(id_type_, id_t, [&] {
      set_unique_keys_and_unique_ids<key_t, id_t>
          <<<grid_size, block_size, 0, stream>>>(
              reinterpret_cast<key_t *>(sorted_keys.data_ptr()),
              reinterpret_cast<uint32_t *>(key_flag_buffer_.data_ptr()),
              reinterpret_cast<key_t *>(unique_keys.data_ptr()),
              reinterpret_cast<id_t *>(unique_key_ids.data_ptr()), num_key_);
    });
  });
  DEMB_CUDA_KERNEL_LAUNCH_CHECK();
}

template <typename T>
__global__ void get_table_range_kernel(
  int64_t num_table,
  int64_t feature_x_batch,
  T const * __restrict__ offsets,
  T const * __restrict__ feature_offsets,
  T * __restrict__ table_range
) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < num_table + 1) {
    T num_feature = feature_offsets[num_table];
    int64_t batch = feature_x_batch / num_feature;
    T feature_offset = feature_offsets[tid];
    T feature_x_batch_offset = feature_offset * batch;
    table_range[tid] = offsets[feature_x_batch_offset];
  }
}

at::Tensor get_table_range(at::Tensor offsets, at::Tensor feature_offsets) {
  if (!offsets.is_cuda()) {
    throw std::runtime_error("Tensor <offsets> must be on CUDA device.");
  }
  if (!feature_offsets.is_cuda()) {
    throw std::runtime_error("Tensor <feature_offsets> must be on CUDA device.");
  }
  int64_t feature_x_batch = offsets.size(0) - 1;
  int64_t num_table = feature_offsets.size(0) - 1;

  auto stream = at::cuda::getCurrentCUDAStream().stream();
  at::Tensor table_range = at::empty_like(feature_offsets);

  int block_size = 128;
  if (num_table + 1 < block_size) {
    block_size = num_table + 1;
  }
  int grid_size = (num_table + block_size) / block_size;
  auto offset_type = scalartype_to_datatype(offsets.dtype().toScalarType());
  DISPATCH_OFFSET_INT_TYPE(offset_type, offset_t, [&] {
    get_table_range_kernel<offset_t><<<grid_size, block_size, 0, stream>>>(
      num_table, feature_x_batch, reinterpret_cast<offset_t*>(offsets.data_ptr()),
      reinterpret_cast<offset_t*>(feature_offsets.data_ptr()),
      reinterpret_cast<offset_t*>(table_range.data_ptr())
    );
  });
  DEMB_CUDA_KERNEL_LAUNCH_CHECK();
  return table_range;
}

std::tuple<at::Tensor, at::Tensor, at::Tensor, at::Tensor>
segmented_unique(at::Tensor keys, at::Tensor segment_range, std::shared_ptr<dyn_emb::UniqueOpBase> unique_op) {

  auto stream = at::cuda::getCurrentCUDAStream().stream();
  int64_t num_total = keys.size(0);
  size_t unique_op_capacity = unique_op->get_capacity();
  if (num_total * 2 > unique_op_capacity) {
    at::Tensor new_keys = at::empty({num_total * 2}, keys.options());
    at::Tensor new_vals = at::empty(
        {num_total * 2},
        at::TensorOptions().dtype(at::kLong).device(keys.device()));
    unique_op->reset_capacity(new_keys, new_vals, num_total * 2, stream);
  }

  at::Tensor h_segment_range = at::empty(segment_range.sizes(), segment_range.options().device(at::kCPU).pinned_memory(true));
  h_segment_range.copy_(segment_range, /*non_blocking=*/true);

  int table_num = segment_range.size(0) - 1;
  std::vector<at::Tensor> tmp_unique_indices(table_num);
  for (int i = 0; i < table_num; ++i) {
    tmp_unique_indices[i] = at::empty_like(keys);
  }

  at::Tensor d_unique_nums = at::empty(table_num, segment_range.options());
  at::Tensor d_unique_indices_table_range = at::zeros(table_num + 1, segment_range.options());

  auto unique_num_type = scalartype_to_datatype(
      convertTypeMetaToScalarType(d_unique_nums.dtype()));
  auto unique_offset_type = scalartype_to_datatype(
      convertTypeMetaToScalarType(d_unique_indices_table_range.dtype()));
  auto inverse_idx = at::empty(num_total, segment_range.options());

  // sync for h_segment_range
  AT_CUDA_CHECK(hipStreamSynchronize(stream));

  for (int i = 0; i < table_num; ++i) {
    int64_t indices_begin = h_segment_range[i].item<int64_t>();
    int64_t indices_end = h_segment_range[i+1].item<int64_t>();
    int64_t indices_length = indices_end - indices_begin;

    if (indices_length == 0) {
      DEMB_CUDA_CHECK(hipMemsetAsync(
          reinterpret_cast<int64_t *>(d_unique_nums.data_ptr()) + i, 0,
          sizeof(int64_t), stream));
      dyn_emb::add_offset(d_unique_nums.data_ptr(), d_unique_indices_table_range.data_ptr(),
                          i, unique_num_type, unique_offset_type, stream);
    } else {
      at::Tensor tmp_indices = keys.slice(0, indices_begin, num_total);
      at::Tensor tmp_inverse_idx = inverse_idx.slice(0, indices_begin, num_total);
      at::Tensor tmp_d_unique_num = d_unique_nums.slice(0, i, table_num);

      at::Tensor previous_d_unique_num = d_unique_indices_table_range.slice(0, i, table_num + 1);
      unique_op->unique(tmp_indices, indices_length, tmp_inverse_idx,
                        tmp_unique_indices[i], tmp_d_unique_num, stream,
                        previous_d_unique_num);
      dyn_emb::add_offset(d_unique_nums.data_ptr(), d_unique_indices_table_range.data_ptr(),
                          i, unique_num_type, unique_offset_type, stream);
    }
  }
  
  at::Tensor h_unique_indices_table_range = at::empty(table_num + 1, segment_range.options().device(at::kCPU));
  AT_CUDA_CHECK(hipMemcpyAsync(
      h_unique_indices_table_range.data_ptr(), d_unique_indices_table_range.data_ptr(),
      (d_unique_indices_table_range.size(0)) * d_unique_indices_table_range.element_size(),
      hipMemcpyDeviceToHost, stream));
  AT_CUDA_CHECK(hipStreamSynchronize(stream));

  int64_t unique_embs_offset = 0;
  int64_t num_unique_total = h_unique_indices_table_range[table_num].item<int64_t>();
  at::Tensor unique_keys = at::empty(num_unique_total, keys.options());
  for (int i = 0; i < table_num; ++i) {
    int64_t tmp_unique_num = h_unique_indices_table_range[i+1].item<int64_t>() - h_unique_indices_table_range[i].item<int64_t>();
    if (tmp_unique_num != 0) {
      void *dst_ptr = reinterpret_cast<char *>(unique_keys.data_ptr()) +
                      unique_embs_offset * unique_keys.element_size();
      void *src_ptr = tmp_unique_indices[i].data_ptr();
      size_t copy_size = tmp_unique_num * unique_keys.element_size();
      AT_CUDA_CHECK(hipMemcpyAsync(dst_ptr, src_ptr, copy_size,
                                    hipMemcpyDeviceToDevice, stream));

    }
    unique_embs_offset += tmp_unique_num;
  }
  return std::make_tuple(unique_keys, inverse_idx, d_unique_indices_table_range, h_unique_indices_table_range);
}

void select(at::Tensor flags, at::Tensor inputs, at::Tensor outputs, at::Tensor num_selected) {

  auto stream = at::cuda::getCurrentCUDAStream().stream();
  int64_t num_total = inputs.size(0);
  auto scalar_type = inputs.dtype().toScalarType();
  auto key_type = scalartype_to_datatype(scalar_type);
  auto num_select_iter_type = scalartype_to_datatype(num_selected.dtype().toScalarType());

  DISPATCH_INTEGER_DATATYPE_FUNCTION(key_type, KeyType, [&] {
    DISPATCH_INTEGER_DATATYPE_FUNCTION(num_select_iter_type, NumSelectedIteratorT, [&] {
      select_async<KeyType, NumSelectedIteratorT>(num_total, flags.data_ptr<bool>(), reinterpret_cast<KeyType*>(inputs.data_ptr()),
        reinterpret_cast<KeyType*>(outputs.data_ptr()), reinterpret_cast<NumSelectedIteratorT*>(num_selected.data_ptr()), inputs.device(), stream);
    });
  });
}

void select_index(at::Tensor flags, at::Tensor output_indices, at::Tensor num_selected) {
  auto stream = at::cuda::getCurrentCUDAStream().stream();
  int64_t num_total = output_indices.size(0);
  auto scalar_type = output_indices.dtype().toScalarType();
  auto key_type = scalartype_to_datatype(scalar_type);
  auto num_select_iter_type = scalartype_to_datatype(num_selected.dtype().toScalarType());

  DISPATCH_INTEGER_DATATYPE_FUNCTION(key_type, KeyType, [&] {
    DISPATCH_INTEGER_DATATYPE_FUNCTION(num_select_iter_type, NumSelectedIteratorT, [&] {
      select_index_async<KeyType, NumSelectedIteratorT>(num_total, flags.data_ptr<bool>(), reinterpret_cast<KeyType*>(output_indices.data_ptr()),
        reinterpret_cast<NumSelectedIteratorT*>(num_selected.data_ptr()), output_indices.device(), stream);
    });
  });
}

} // namespace dyn_emb

void bind_index_calculation_op(py::module &m) {
  m.def("get_table_range", &dyn_emb::get_table_range,
    "Make offsets from <feature, batch> scope into <table> scope",
    py::arg("offsets"), py::arg("feature_offsets"));

  m.def("segmented_unique", &dyn_emb::segmented_unique,
    "Dose segmented unique operation on keys with segment_range, return tuple<unique_keys, inverse, unique_keys_table_range, h_unique_keys_table_range>",
    py::arg("keys"), py::arg("segment_range"), py::arg("unique_op"));
  
  m.def(
    "select", &dyn_emb::select,
    "Select items in inputs which flags are true.", 
    py::arg("flags"), py::arg("inputs"), py::arg("outputs"), py::arg("num_selected")
  );
  m.def(
    "select_index", &dyn_emb::select_index,
    "Select items' indices where flags are true.", 
    py::arg("flags"), py::arg("output_indices"), py::arg("num_selected")
  );
}
